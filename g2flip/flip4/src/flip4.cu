#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
//#include "windows.h"
#include <time.h>
#include <sys/time.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include "mapper.h"

// gtx1080
// Iterations	Games per block		Grid size	Latency		Mega moves per sec
// 2000			8					256			0.159 ms	772-1200

#define ITERATIONS 2000

// Early break in loops increases throughput by 10% even though threads gets more diverted
#define earlybreak 1

// Use float gains 10%
#define usefloat 1

// Copy result after each game iteration
#define COPY_RESULT_TO_HOST 1

// Print transcript from one of the played games
//#define DO_TRANSCRIPT 1

#define GAMES_PER_BLOCK 8
#define GRID_SIZE 256

#define DEFAULT_DEPTH 60

#define MAX_STREAMS 2
#define MAX_DEVICES 1

// Parameters used for switching between float and int for counters
#ifdef usefloat
#define zero 0.0
#define one 1.0
#define three 3.0
#define five 5.0
#define six 6.0
#define eight 8.0
#define sixtyfour 64.0
#define counter float
#else
#define zero 0
#define one 1
#define three 3
#define five 5
#define six 6
#define eight 8
#define sixtyfour 64
#define counter int
#endif

#define IS_MASTER if (threadIdx.x % 8 == 1)

// Left edge
const uint64_t NotA1A8 = 18374403900871474942ULL;
// Right edge
const uint64_t NotH1H8 = 9187201950435737471ULL;
// No side edges
const uint64_t NotEDGE = NotH1H8 & NotA1A8;
__constant__ const int DIRECTIONS[] = { -9, -8, -7, -1, 1, 7, 8, 9 };


void initPositions(ulonglong2* p,int numberOfPositions);

uint64_t getBoard(char* board, char c);
int getRow(char* s, char c);
char* toBoard(ulonglong2 b, int labels, int color, char* board);
char* toRow(unsigned int r, int color, char* row);
char* toBoard_Pattern(uint64_t b, int labels, char* board);
char* formatTranscript(int* transcript, int labels, int color, char* board);
int readFromFile(char *filename, ulonglong2* positions, int maxPositions);

long long getCurrentTs(){
	struct timeval tp;
    gettimeofday(&tp, NULL);
    long long ms = (long long) tp.tv_sec * 1000L + tp.tv_usec / 1000;
    return ms;
}

// Overall configuration of execution
int streams = 1;
int depth = DEFAULT_DEPTH;
int initFromFile = 0;

double statistics[MAX_DEVICES];

// TODO:
// implement node structure
// refactor so takeback is possible
// Count disc differance at end of game. Store value in node. 
// Use alpha-beta pruning
// multi-threaded s� begge devices anvendes
// fix fejl ved flere streams
// 

void onErrorExit(char *msg, hipError_t error, int line){
	if (error != hipSuccess){
		printf("%s cuda error code %d, line(%d): %s\n", msg, error, line, hipGetErrorString(error));   
		exit(EXIT_FAILURE); 
	}
}

/************************************************************************************
 *
 *	Utility methods for calculating lines excluding H1-H8 and A1-A8.
 *  Only invoked once
 *
 ************************************************************************************/
uint64_t getNotH1H8(){
	uint64_t v = 0LL;
	for(int y = 0; y < 64; y+=8){
		v |= (1LL<< y); 
	} 
	return ~v;
}

uint64_t getNotA1A8(){
	uint64_t v = 0LL;
	for (int y = 0; y < 64; y += 8){
		v |= (128LL<< y); 
	} 
	return ~v;
}

__device__ __inline__ uint64_t getLegalMoves(int direction, uint64_t e, uint64_t me, uint64_t op){
	uint64_t v;		// temp variable
	uint64_t xor_pattern = 0ULL;

	int d = abs(direction);
	v = me;
	if (direction < 0) {
		v <<= d;
	}else{
		v >>= d;
	}
	if (d != 8) v &= NotEDGE;
#pragma __unroll
	for (int a = 0; a < 6; a++){
		v &= op;
#ifdef earlybreak
		if (v == 0) break;
#endif
		if (direction < 0) {
			v <<= d;
		}
		else{
			v >>= d;
		}

		xor_pattern |= (e & v);
		if (d != 8) v &= NotEDGE;
	}
	return xor_pattern;
}

/************************************************************************************
 *
 *	Find legal moves and store them in options array
 *
 ************************************************************************************/
__device__ void getLegalAllMoves(ulonglong2 *board, int isWhiteToPlay, uint64_t *options){
	__shared__ uint64_t me[GAMES_PER_BLOCK];
	__shared__ uint64_t op[GAMES_PER_BLOCK];
	__shared__ uint64_t e[GAMES_PER_BLOCK];

	IS_MASTER{
		if (isWhiteToPlay){
			me[threadIdx.y] = board[0].y;
			op[threadIdx.y] = board[0].x;
		}
		else{
			op[threadIdx.y] = board[0].y;
			me[threadIdx.y] = board[0].x;
		}
		e[threadIdx.y] = ~(me[threadIdx.y] | op[threadIdx.y]);
	}
	__syncthreads();

	int a = threadIdx.x % 8;
	options[a] = getLegalMoves(DIRECTIONS[a], e[threadIdx.y], me[threadIdx.y], op[threadIdx.y]);
}

/************************************************************************************
 *
 * Select option to play. Return bit nr to identify location
 *
 ************************************************************************************/
__device__ __inline__ int selectOption(uint64_t options){

	// TODO: Optimize and do select by pruning etc.
	// Select the middle options of the possible options	
	counter numberOfOptions = (counter) __popcll(options);
	if (numberOfOptions > three){
#ifdef usefloat
		numberOfOptions /= 2.0;
#else
		numberOfOptions >>= 1;
#endif		
		for (counter a = zero; a<numberOfOptions; a++){
			int optionDeselected = __ffsll(options);
			options ^= (1ULL << (optionDeselected - 1));
		}
	}
	return __ffsll(options);
}



__device__ __inline__ uint64_t flip(int direction, uint64_t location, uint64_t option, uint64_t boardOpp){
	uint64_t xorPattern = 0ULL;
	uint64_t d = abs(direction);
	if (option & location){
		uint64_t v = location;
#pragma __unroll
		for (int a = 0; a < 6; a++){
			if (direction > 0) {
				v <<= d;
			}
			else{
				v >>= d;
			}
			if (!(boardOpp & v)) break;
			xorPattern ^= v;
		}
	}
	return xorPattern;
}


__device__ int makeMove(int globalIdx, ulonglong2* positions, ulonglong2* result, int isWhiteToPlay, uint64_t *options){
	__shared__ uint64_t location[GAMES_PER_BLOCK];
	__shared__ uint64_t boardOpp[GAMES_PER_BLOCK];
	__shared__ uint64_t xorPattern[GAMES_PER_BLOCK][8];
	__shared__ int bitNr;

	IS_MASTER{
		bitNr = 0;
		location[threadIdx.y] = 0ULL;
		for (int a = 0; a < 8; a++){
			location[threadIdx.y] |= options[a];
		}

		if (location[threadIdx.y] != 0ULL){
			if (isWhiteToPlay){
				boardOpp[threadIdx.y] = positions[globalIdx].x;
			}
			else{
				boardOpp[threadIdx.y] = positions[globalIdx].y;
			}

			bitNr = selectOption(location[threadIdx.y]);
			if (bitNr > 0){
				location[threadIdx.y] = 1ULL << (bitNr-1);
			}
		}
	}
	__syncthreads();
	if (location[threadIdx.y] == 0ULL) return 0;

	int a = threadIdx.x % 8;
	xorPattern[threadIdx.y][a] = flip(DIRECTIONS[a], location[threadIdx.y], options[a], boardOpp[threadIdx.y]);
	__syncthreads();

	IS_MASTER{
		uint64_t pattern = 0ULL;
		for (int b = 0; b < 8; b++){
			pattern |= xorPattern[threadIdx.y][b];
		}
		result[globalIdx].x = positions[globalIdx].x ^ pattern;
		result[globalIdx].y = positions[globalIdx].y ^ pattern;

		if (isWhiteToPlay){
			result[globalIdx].y |= location[threadIdx.y];
		}
		else{
			result[globalIdx].x |= location[threadIdx.y];
		}
	}
	return bitNr;
}

__device__ __inline__ void swap(ulonglong2* p1, ulonglong2* p2){
	ulonglong2* temp = p1;
	p1 = p2;
	p2 = temp;
}

/************************************************************************************
 *
 * TODO: count disc differance at end of game. Store value in node. Use alpha-beta pruning
 *
 ************************************************************************************/
__global__ void play(ulonglong2* positions, ulonglong2* result, int isWhiteToPlayParm, counter depth, int* transcript)
{	  
	__shared__ counter endOfGame[GAMES_PER_BLOCK];
	__shared__ int isWhiteToPlay[GAMES_PER_BLOCK];
	__shared__ int globalIdx[GAMES_PER_BLOCK];
	__shared__ uint64_t options[GAMES_PER_BLOCK][8];
	__shared__ int movenr[GAMES_PER_BLOCK];

	IS_MASTER{
		endOfGame[threadIdx.y] = zero;
		isWhiteToPlay[threadIdx.y] = isWhiteToPlayParm;
		globalIdx[threadIdx.y] = blockIdx.x * GAMES_PER_BLOCK + threadIdx.y;
		movenr[threadIdx.y] = 0;
	}
	__syncthreads();

	do{
		getLegalAllMoves(&positions[globalIdx[threadIdx.y]], isWhiteToPlay[threadIdx.y], options[threadIdx.y]);
		__syncthreads();
		int chosenMove = makeMove(globalIdx[threadIdx.y], positions, result, isWhiteToPlay[threadIdx.y], options[threadIdx.y]);

		IS_MASTER{
			isWhiteToPlay[threadIdx.y] ^= 1;
			if (chosenMove != 0){
				positions[globalIdx[threadIdx.y]] = result[globalIdx[threadIdx.y]];
				endOfGame[threadIdx.y] = zero;
#ifdef DO_TRANSCRIPT
				transcript[chosenMove - 1] = movenr + 1;
#endif
				movenr[threadIdx.y]++;
			}
			else{
				// pass
				endOfGame[threadIdx.y]++;
			}
		}
		__syncthreads();
		if (endOfGame[threadIdx.y] > one) break;
	} while (movenr[threadIdx.y] < depth);
	__syncthreads();
	// Copy options to result
	/*
	IS_MASTER{
		for (int a = 0; a < 8; a++){
			result[globalIdx].x |= options[a];
			result[globalIdx].y |= options[a];
		}
	}
	*/
	__syncthreads();
	//int diffForBlack = __popcll(positions[globalIdx].y) - __popcll(positions[globalIdx].x);

}


/************************************************************************************
 *
 * Kernel starter
 *
 ************************************************************************************/
int testKernel(int streams, int device, int depth, int initFromFile)
{	
	int block_size = 8 * GAMES_PER_BLOCK;
    dim3 grid(GRID_SIZE);
    dim3 threads(8, GAMES_PER_BLOCK);

    hipError_t error;
	hipStream_t stream[MAX_STREAMS];
		
	int device_count;
	error = hipGetDeviceCount(&device_count);
	onErrorExit("getDeviceCount", error, __LINE__);
			
	error = hipSetDevice(device);
	onErrorExit("setDevice", error, __LINE__);

	error = hipGetDevice(&device); 
	onErrorExit("setDevice", error, __LINE__);
	printf("device %d \n", device);	

	for(int i=0; i < streams; i++){
		error = hipStreamCreate(&stream[i]);
		onErrorExit("stream create", error, __LINE__);
	}
	
    int numberOfPositions = GAMES_PER_BLOCK * GRID_SIZE;
	int streamSize = numberOfPositions;

	//int area = sizeof(ulonglong2) * streamSize;
	//printf("Allocating host memory x %d: %d b\n", streams, area);
	ulonglong2* hPositions[MAX_STREAMS];
	ulonglong2* dPositions[MAX_STREAMS];
	ulonglong2* hFlipped[MAX_STREAMS];
	ulonglong2* dFlipped[MAX_STREAMS];
	int* hTranscript[MAX_STREAMS];
	int* dTranscript[MAX_STREAMS];

	// Allocate host and device memory

	for(int i = 0; i < streams; i++){
		//printf("Alloc mem... stream %d\n",i);
		error = hipHostMalloc((void **) &hPositions[i], sizeof(ulonglong2) * streamSize);		
		onErrorExit("memory", error, __LINE__);
		memset(hPositions[i], 0, sizeof(ulonglong2) * streamSize);
		
		error = hipHostMalloc((void **) &hFlipped[i], sizeof(ulonglong2) * streamSize);
		onErrorExit("memory", error, __LINE__);
		memset(hFlipped[i], 0, sizeof(ulonglong2) * streamSize);

		error = hipHostMalloc((void **)&hTranscript[i], sizeof(int) * streamSize * 64);
		onErrorExit("memory", error, __LINE__);
		memset(hTranscript[i], 0, sizeof(int) * streamSize * 64);

		//printf("Init positions... stream %d\n",i);
		if (initFromFile != 0){
			int loaded = readFromFile("games.txt", hPositions[i], numberOfPositions);
			printf("Loaded %d positions\n", loaded);
		}
		else{
			initPositions(hPositions[i], streamSize);
		}

		char preBoard[4000];
//		for(int g=0; g< numberOfPositions; g++){
//			preBoard[0] = 0;
//			printf("Before %d\n%s\n", g, toBoard(hPositions[0][g], 1, 0, preBoard));
//		}

		error = hipMalloc((void **) &dPositions[i], sizeof(ulonglong2) * streamSize);
		onErrorExit("memory", error, __LINE__);

		error = hipMalloc((void **) &dFlipped[i], sizeof(ulonglong2) * streamSize);
		onErrorExit("memory", error, __LINE__);

		error = hipMalloc((void **)&dTranscript[i], sizeof(int) * streamSize * 64);
		onErrorExit("memory", error, __LINE__);

	}    

    //printf("Invoke CUDA Kernel...\n");
	long long numberOfMoves = (long long) numberOfPositions * depth * ITERATIONS;
	printf("Number of moves: %5.3f giga moves\n",  numberOfMoves / 1000000000.0);

	counter maxDepth = (counter) depth;

#ifdef WIN
	SYSTEMTIME startTime;
	GetSystemTime(&startTime);
	printf("Start: %ld:%ld.%ld\n", startTime.wMinute, startTime.wSecond, startTime.wMilliseconds);
	long long startMs = (long long)(startTime.wMinute * 60 * 1000 + startTime.wSecond * 1000 + startTime.wMilliseconds);
#else
	time_t startTime = time(NULL);
	printf("Start: ");
	printf(ctime(&startTime));
	long long startMs = getCurrentTs();
#endif
	for(int z = 0; z< ITERATIONS; z++){		
		int i = z % streams;		
		
		// copy host memory to device	
		error = hipMemcpyAsync(dPositions[i], hPositions[i], sizeof(ulonglong2) * streamSize, hipMemcpyHostToDevice, stream[i]);
		
		onErrorExit("memory", error, __LINE__);
		hipStreamSynchronize(stream[i]);

		play<<< grid, threads, 0, stream[i] >>>(dPositions[i], dFlipped[i], 0, maxDepth, dTranscript[i]);

		// Copy result from device to host		
#ifdef DO_TRANSCRIPT
		// FIX-ME: Won't work with multiple streams...
		error = hipMemcpyAsync(&hTranscript[i][0], &dTranscript[i][0], sizeof(int) * streamSize * 64, hipMemcpyDeviceToHost, stream[i]);
		onErrorExit("memory", error, __LINE__);
#endif

#ifdef COPY_RESULT_TO_HOST
		// Copy only xor patterns
		error = hipMemcpyAsync(hFlipped[i], dFlipped[i], sizeof(ulonglong2) * streamSize, hipMemcpyDeviceToHost, stream[i]);
		onErrorExit("memory", error, __LINE__);				
#endif
	}
	for(int i=0; i < streams; i++){
		hipStreamSynchronize(stream[i]);
	}	

#ifdef WIN
	GetSystemTime(&startTime);
	long long endTime = (long long)(startTime.wMinute * 60 * 1000 + startTime.wSecond * 1000 + startTime.wMilliseconds);
	long long endMs = endTime - startMs;
#else
	time_t endTime = time(NULL);
	long long endMs = getCurrentTs() - startMs;
#endif

	char board[4000];
	for(int i=0; i < streams; i++){
		// Prepare print-buffer// Prepare print-buffer
		board[0] = 0;
		printf("Before %d\n%s\n", i, toBoard(hPositions[i][10], 1, 0, board));

		// Prepare print-buffer		
		board[0] = 0;
		printf("After %d\n%s\n", i, toBoard(hFlipped[i][10], 1, 0, board));
//		for(int g=0; g < numberOfPositions; g++){
//			board[0] = 0;
//			printf("After %d\n%s\n", g, toBoard(hFlipped[i][g], 1, 0, board));
//		}
#ifdef DO_TRANSCRIPT
		// Prepare print-buffer		
		board[0] = 0;
		printf("Transcript %d:\n%s\n", i, formatTranscript(hTranscript[i], 1, 0, board));		
//		printf("After %d\n%s\n", i, toBoard_Pattern(hFlipped[i][0], 1, board));		
#endif
	}

//	for(int i=0; i < numberOfPositions ; i++){
//		// printf("%d: I=%I64U,%I64U R=%I64U,%I64U \n", i , hPositions[0][i].x, hPositions[0][i].y, );
//		board[0] = 0;
//		printf("%d\n%s\n", i, toBoard(hFlipped[0][i], 1, 0, board));
//	}
	
#ifdef WIN
	printf("Thread end:   %ld:%ld.%ld\n", time.wMinute, time.wSecond, time.wMilliseconds);
#else
	printf("Thread end: ");
	printf(ctime(&endTime));
#endif
	if (endMs<1){
		printf("Warning: Duration < 1 ms\n");
		endMs = 1;
	}
	double megaMovesPerSecond = ((double)(numberOfMoves / endMs)) / 1000.0;
	printf("Thread total: %ld ms\nThread: %5.3f mega moves per second\n", endMs, megaMovesPerSecond);

	double gigaMovesPerSecond = ((double)(numberOfMoves / endMs)) / 1000.0 / 1000.0;
	printf("Thread total: %ld ms\nThread: %5.3f giga moves per second\n", endMs, gigaMovesPerSecond);


    // Clean up memory
	for(int i=0; i < streams; i++){
		error = hipHostFree(hPositions[i]);
		onErrorExit("memory", error, __LINE__);
		error = hipHostFree(hFlipped[i]);
		onErrorExit("memory", error, __LINE__);
		error = hipFree(dPositions[i]);
		onErrorExit("memory", error, __LINE__);
		error = hipFree(dFlipped[i]);
		onErrorExit("memory", error, __LINE__);
		error = hipFree(dTranscript[i]);
		onErrorExit("memory", error, __LINE__);
		error = hipStreamDestroy(stream[i]);
		onErrorExit("destroy stream", error, __LINE__);
	}

    hipDeviceReset();
	statistics[device] = megaMovesPerSecond;
	return 0;
}

//
//DWORD WINAPI testKernalThreadable(LPVOID lpParam){
//	return testKernel(streams, *(int*) lpParam, depth, initFromFile);
//}

/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Othello Using CUDA] - Starting...\n");    

	//HANDLE threadHandles[MAX_DEVICES];

	int devices = 1;
	int nowait = 0;
	int device = 0;

	for (int i = 1; i < argc; i++){
		if (strncmp(argv[i],"-Dstreams", 9)==0){
			streams = atoi(argv[++i]);
		}
		if (strncmp(argv[i],"-Ddevices", 9)==0){
			devices = atoi(argv[++i]);
			if (devices > MAX_DEVICES){
				printf("Maximum number of devices is %d", MAX_DEVICES);
				exit(1);
			}
		}else if (strncmp(argv[i],"-Ddevice", 8)==0){
			device = atoi(argv[++i]);
		}
		if (strncmp(argv[i],"-Ddepth", 7)==0){
			depth = atoi(argv[++i]);
		}
		if (strncmp(argv[i],"-Dnowait", 7)==0){
			nowait = 1;
		}
		if (strncmp(argv[i], "-Dfile", 6) == 0){
			initFromFile = 1;
		}
	}

	int result = testKernel(streams, device, depth, initFromFile);


//	int deviceNr[MAX_DEVICES];
//
//	for (int i = 0; i < devices; i++){
//		deviceNr[i] = i;
//		threadHandles[i] = CreateThread(NULL, 0, testKernalThreadable, &deviceNr[i], 0, NULL);
//	}
//
//	WaitForMultipleObjects(devices, threadHandles, TRUE, INFINITE);
//
//	double mmps = 0.0;
//	for (int i = 0; i < devices; i++){
//		CloseHandle(threadHandles[i]);
//		mmps += statistics[i];
//	}
//	printf("Total: %5.3f mega moves per second\n", mmps);

	if (!nowait){
//		getchar();
	}
    exit(0);
}
